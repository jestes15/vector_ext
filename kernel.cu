﻿#include <vector>
#include <iostream>
#include <chrono>

#define USE_CUDA
#include "vector_ext.cuh"

template <typename T>
void print_array(std::vector<T> &array) {
    std::cout << "[ ";
    for (auto &i : array) {
        std::cout << i << " ";
    }
    std::cout << "]" << std::endl;
}

template <typename T>
int validate_dest(std_vec::vector_ext<T> &dest, std_vec::vector_ext<T> &src, std_vec::vector_ext<T> &src2) {
    for (int i = 0; i < dest.size(); i++) {
        if (dest[i] != src[i] + src2[i]) {
            std::cout << "Error at index " << i << std::endl;
            return 1;
        }
    }
    return 0;
}

int main()
{
    long size = 6000000;
    std_vec::vector_ext<int> src1(size), src2(size), dest(size);

    for (auto i = 0; i < size; ++i) {
        src1.push_back(i);
        src2.push_back(i);
    }

    auto start_for = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    for (long i = 0; i < size; ++i) {
        dest[i] = src1[i] + src2[i];
    }
    auto end_for = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    std::cout << "Time taken: " << (end_for - start_for) << "ms" << std::endl;


    auto start = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    dest = src1 + src2;
    auto stop = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    std::cout << "Time taken: " << (stop - start) << "ms" << std::endl;

    if (validate_dest(dest, src1, src2)) {
        return 1;
    }

    std_vec::vector_ext<int> src(20);
    src.generate_random_list_cuda(10000);

    print_array(src);

    return 0;
}