﻿#include <vector>
#include <iostream>
#include <chrono>

#define USE_CUDA
#include "vector_ext.cuh"

template <typename T>
void print_array(std::vector<T> &array)
{
    std::cout << "[ ";
    for (auto &i : array)
    {
        std::cout << i << " ";
    }
    std::cout << "]" << std::endl;
}

template <typename T>
int validate_dest(std_vec::vector_ext<T> &dest, std_vec::vector_ext<T> &src, std_vec::vector_ext<T> &src2)
{
    for (int i = 0; i < dest.size(); i++)
    {
        if (dest[i] != src[i] + src2[i])
        {
            std::cout << "Error at index " << i << std::endl;
            return 1;
        }
    }
    return 0;
}
namespace tests
{
    #if defined(MATRIX_MUL)
    // TODO: Finish test for user_space::matrix_mul
    void _test()
    {
        const int size = 10;

        auto src = new int [size][size];
        auto src2 = new int [size][size];
        auto dest = new int [size][size];

        std::random_device gen;
        std::uniform_int_distribution<int> dist(-300, 300);

        std::for_each_n(src, size, [&dist, &gen](int *row) {
            std::for_each_n(row, size, [&dist, &gen](int &i) {
                i = static_cast<int>(dist(gen));
            });
        });

        std::for_each_n(src2, size, [&dist, &gen](int *row) {
            std::for_each_n(row, size, [&dist, &gen](int &i) {
                i = static_cast<int>(dist(gen));
            });
        });

        std::for_each_n(src, size, [](int *row) {
            std::for_each_n(row, size, [](int &i) {
                std::cout << i << " ";
            });
            std::cout << std::endl;
        });

        user_space::matrix_mul(dest, src, src2, size, size, size, size);

        std::for_each_n(dest, size, [](int *row) {
            std::for_each_n(row, size, [](int &i) {
                std::cout << i << " ";
            });
            std::cout << std::endl;
        });

    }
    #endif

    int t()
    {
        long size = 6000000;
        std_vec::vector_ext<int> src1(size), src2(size), dest(size);

        for (auto i = 0; i < size; ++i)
        {
            src1.push_back(i);
            src2.push_back(i);
        }

        auto start_for = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        for (long i = 0; i < size; ++i)
        {
            dest[i] = src1[i] + src2[i];
        }
        auto end_for = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        std::cout << "Time taken: " << (end_for - start_for) << "ms" << std::endl;

        auto start = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        dest = src1 + src2;
        auto stop = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

        std::cout << "Time taken: " << (stop - start) << "ms" << std::endl;

        if (validate_dest(dest, src1, src2))
        {
            return 1;
        }

        std_vec::vector_ext<int> src(20);
        src.generate_random_list_cuda(10000);

        print_array(src);

        return 0;
    }
}

int main()
{
    #if defined(MATRIX_MUL)
    tests::_test();
    #endif

    return 0;
}