﻿#include "kernel_impl.cuh"

#include "types.cuh"

#include <iostream>
#include <map>
#include <optional>
#include <sstream>

namespace test_kernel_implementations
{
namespace test_kernel_implementations_enum
{
enum tests
{
    add_user_space,
    sub_user_space,
    mul_user_space,
    div_user_space,
    generate_random_number_user_space,
    generate_random_number_user_space_with_seed,
    matrix_multiply_user_space,
    matrix_multiply_user_space_with_squished_matrix,
};

std::map<tests, std::string> test_names = {
    {add_user_space, "add_user_space"},
    {sub_user_space, "sub_user_space"},
    {mul_user_space, "mul_user_space"},
    {div_user_space, "div_user_space"},
    {generate_random_number_user_space, "generate_random_number_user_space"},
    {generate_random_number_user_space_with_seed, "generate_random_number_user_space_with_seed"},
    {matrix_multiply_user_space, "matrix_multiply_user_space"},
    {matrix_multiply_user_space_with_squished_matrix, "matrix_multiply_user_space_with_squished_matrix"},
};
} // namespace test_kernel_implementations_enum

class test_kernel_implementation_class
{
  public:
    test_kernel_implementation_class() = default;
    ~test_kernel_implementation_class() = default;

    std::map<test_kernel_implementations_enum::tests, std::optional<std::string>> run()
    {
        test_add_user_space();
        test_sub_user_space();
        test_mul_user_space();
        test_div_user_space();

        test_generate_random_number_user_space();
        test_generate_random_number_user_space_with_seed();
        
        test_matrix_multiply_user_space();
        test_matrix_multiply_user_space_with_squished_matrix();

        return results;
    }

  private:
    std::map<test_kernel_implementations_enum::tests, std::optional<std::string>> results;

    void test_add_user_space()
    {
        std::cout << "test_add_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_add_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::add_user_space] = ss.str();
    }

    void test_sub_user_space()
    {
        std::cout << "test_sub_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_sub_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::sub_user_space] = ss.str();
    }

    void test_mul_user_space()
    {
        std::cout << "test_mul_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_mul_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::mul_user_space] = ss.str();
    }

    void test_div_user_space()
    {
        std::cout << "test_div_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_div_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::div_user_space] = ss.str();
    }

    void test_generate_random_number_user_space()
    {
        std::cout << "test_generate_random_number_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_generate_random_number_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::generate_random_number_user_space] = ss.str();
    }

    void test_generate_random_number_user_space_with_seed()
    {
        std::cout << "test_generate_random_number_user_space_with_seed" << std::endl;
        std::stringstream ss;
        ss << "test_generate_random_number_user_space_with_seed" << std::endl;
        results[test_kernel_implementations_enum::tests::generate_random_number_user_space_with_seed] = ss.str();
    }

    void test_matrix_multiply_user_space()
    {
        std::cout << "test_matrix_multiply_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_matrix_multiply_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::matrix_multiply_user_space] = ss.str();
    }

    void test_matrix_multiply_user_space_with_squished_matrix()
    {
        std::cout << "test_matrix_multiply_user_space_with_squished_matrix" << std::endl;
        std::stringstream ss;
        ss << "test_matrix_multiply_user_space_with_squished_matrix" << std::endl;
        results[test_kernel_implementations_enum::tests::matrix_multiply_user_space_with_squished_matrix] = ss.str();
    }
};
} // namespace test_kernel_implementations

int main()
{

}