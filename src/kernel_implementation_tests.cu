﻿#include "kernel_impl.cuh"

#include "types.cuh"

#include <iostream>
#include <map>
#include <optional>
#include <sstream>

namespace test_kernel_implementations
{
namespace test_kernel_implementations_enum
{
enum tests
{
    add_user_space,
    sub_user_space,
    mul_user_space,
    div_user_space,
    generate_random_number_user_space,
    generate_random_number_user_space_with_seed,
    matrix_multiply_user_space,
    matrix_multiply_user_space_with_squished_matrix,
};

std::map<tests, std::string> test_names = {
    {add_user_space, "add_user_space"},
    {sub_user_space, "sub_user_space"},
    {mul_user_space, "mul_user_space"},
    {div_user_space, "div_user_space"},
    {generate_random_number_user_space, "generate_random_number_user_space"},
    {generate_random_number_user_space_with_seed, "generate_random_number_user_space_with_seed"},
    {matrix_multiply_user_space, "matrix_multiply_user_space"},
    {matrix_multiply_user_space_with_squished_matrix, "matrix_multiply_user_space_with_squished_matrix"},
};
} // namespace test_kernel_implementations_enum

class test_kernel_implementation_class
{
  public:
    test_kernel_implementation_class() = default;
    ~test_kernel_implementation_class() = default;

    std::map<test_kernel_implementations_enum::tests, std::optional<std::string>> run()
    {
        test_add_user_space();
        test_sub_user_space();
        test_mul_user_space();
        test_div_user_space();

        test_generate_random_number_user_space();
        test_generate_random_number_user_space_with_seed();

        test_matrix_multiply_user_space();
        test_matrix_multiply_user_space_with_squished_matrix();

        return results;
    }

  private:
    std::map<test_kernel_implementations_enum::tests, std::optional<std::string>> results;

    bool check_1d_array_result(int *expected_result, int *result, int size)
    {
        for (int i = 0; i < size; i++)
        {
            if (expected_result[i] != result[i])
            {
                return false;
            }
        }
        return true;
    }

    template <typename _Type, std::size_t size>
    bool check_1d_array_result(std::array<_Type, size> expected_result, std::array<_Type, size> result)
    {
        for (int i = 0; i < size; i++)
        {
            if (expected_result[i] != result[i])
                return false;
        }
        return true;
    }

    std::string print_1d_array(int *array, int size)
    {
        std::stringstream ss;
        ss << "[";
        for (int i = 0; i < size; i++)
        {
            ss << array[i];
            if (i != size - 1)
            {
                ss << ", ";
            }
        }
        ss << "]";

        return ss.str();
    }

    template <typename _Type, std::size_t size> std::string print_1d_array(std::array<_Type, size> array)
    {
        std::stringstream ss;
        ss << "[";
        for (int i = 0; i < size; i++)
        {
            ss << array[i];
            if (i != size - 1)
            {
                ss << ", ";
            }
        }
        ss << "]";

        return ss.str();
    }

    void test_add_user_space()
    {
        std::array<int, 5> left_array = {1, 2, 3, 4, 5};
        std::array<int, 5> right_array = {1, 2, 3, 4, 5};

        std::array<int, 5> result;
        std::array<int, 5> expected_result = {2, 4, 6, 8, 10};

        std_vec::user_space::add(result, left_array, right_array);

        if (check_1d_array_result(expected_result, result))
            this->results[test_kernel_implementations_enum::tests::add_user_space] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Result of addition between " << print_1d_array(left_array) << " and " << print_1d_array(right_array)
               << " is " << print_1d_array(result) << " but expected " << print_1d_array(expected_result) << std::endl;
            this->results[test_kernel_implementations_enum::tests::add_user_space] = ss.str();
        }
    }
    void test_sub_user_space()
    {
        std::array<int, 5> left_array = {1, 2, 3, 4, 5};
        std::array<int, 5> right_array = {1, 2, 3, 4, 5};

        std::array<int, 5> result;
        std::array<int, 5> expected_result = {0, 0, 0, 0, 0};

        std_vec::user_space::sub(result, left_array, right_array);

        if (check_1d_array_result(expected_result, result))
            this->results[test_kernel_implementations_enum::tests::sub_user_space] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Result of subtraction between " << print_1d_array(left_array) << " and "
               << print_1d_array(right_array) << " is " << print_1d_array(result) << " but expected "
               << print_1d_array(expected_result) << std::endl;
            this->results[test_kernel_implementations_enum::tests::sub_user_space] = ss.str();
        }
    }
    void test_mul_user_space()
    {
        std::array<int, 5> left_array = {1, 2, 3, 4, 5};
        std::array<int, 5> right_array = {1, 2, 3, 4, 5};

        std::array<int, 5> result;
        std::array<int, 5> expected_result = {1, 4, 9, 16, 25};

        std_vec::user_space::mul(result, left_array, right_array);

        if (check_1d_array_result(expected_result, result))
            this->results[test_kernel_implementations_enum::tests::mul_user_space] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Result of addition between " << print_1d_array(left_array) << " and " << print_1d_array(right_array)
               << " is " << print_1d_array(result) << " but expected " << print_1d_array(expected_result) << std::endl;
            this->results[test_kernel_implementations_enum::tests::mul_user_space] = ss.str();
        }
    }
    void test_div_user_space()
    {
        std::array<int, 5> left_array = {1, 2, 3, 4, 5};
        std::array<int, 5> right_array = {1, 2, 3, 4, 5};

        std::array<int, 5> result;
        std::array<int, 5> expected_result = {1, 1, 1, 1, 1};

        std_vec::user_space::div(result, left_array, right_array);

        if (check_1d_array_result(expected_result, result))
            this->results[test_kernel_implementations_enum::tests::div_user_space] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Result of addition between " << print_1d_array(left_array) << " and " << print_1d_array(right_array)
               << " is " << print_1d_array(result) << " but expected " << print_1d_array(expected_result) << std::endl;
            this->results[test_kernel_implementations_enum::tests::div_user_space] = ss.str();
        }
    }

    void test_generate_random_number_user_space()
    {
        std::array<int, 5> first_array = {};
        std::array<int, 5> second_array = {};

        std_vec::user_space::generate_random_number(first_array, 5, 10);
        std_vec::user_space::generate_random_number(second_array, 5, 10);

        if (!check_1d_array_result(first_array, second_array))
            this->results[test_kernel_implementations_enum::tests::div_user_space] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Arrays " << print_1d_array(first_array) << " and " << print_1d_array(second_array)
               << " are the same " << std::endl;
            this->results[test_kernel_implementations_enum::tests::div_user_space] = ss.str();
        }
    }

    void test_generate_random_number_user_space_with_seed()
    {
        // std::cout << "test_generate_random_number_user_space_with_seed" << std::endl;
        std::stringstream ss;
        ss << "test_generate_random_number_user_space_with_seed" << std::endl;
        results[test_kernel_implementations_enum::tests::generate_random_number_user_space_with_seed] = ss.str();
    }

    void test_matrix_multiply_user_space()
    {
        // std::cout << "test_matrix_multiply_user_space" << std::endl;
        std::stringstream ss;
        ss << "test_matrix_multiply_user_space" << std::endl;
        results[test_kernel_implementations_enum::tests::matrix_multiply_user_space] = ss.str();
    }

    void test_matrix_multiply_user_space_with_squished_matrix()
    {
        // std::cout << "test_matrix_multiply_user_space_with_squished_matrix" << std::endl;
        std::stringstream ss;
        ss << "test_matrix_multiply_user_space_with_squished_matrix" << std::endl;
        results[test_kernel_implementations_enum::tests::matrix_multiply_user_space_with_squished_matrix] = ss.str();
    }
};
} // namespace test_kernel_implementations

int main()
{
    test_kernel_implementations::test_kernel_implementation_class test_kernel_implementation;
    auto results = test_kernel_implementation.run();

    for (auto const &result : results)
    {
        if (result.second.has_value())
            std::cout << "\033[31mTest "
                      << test_kernel_implementations::test_kernel_implementations_enum::test_names[result.first]
                      << " failed: " << result.second.value() << "\033[0m" << std::endl;
        else
        {
            std::cout << "\033[32mTest "
                      << test_kernel_implementations::test_kernel_implementations_enum::test_names[result.first]
                      << " passed\033[0m" << std::endl;
        }
    }
}