#include "hip/hip_runtime.h"
#include "quaternion.cuh"
#include "types.cuh"

#include <iostream>
#include <sstream>
#include <optional>
#include <unordered_map>

namespace test_quaternion
{
namespace test_quaternion_enum
{
enum tests
{
    default_constructor,
    four_parameter_constructor,
    three_parameter_constructor,
    two_parameter_constructor,
    one_parameter_constructor,
    unit_vector_constructor,
    array_constructor,
    addition,
    subtraction,
    multiplication,
    scalar_multiplication,
    conjugate,
    norm,
    inverse,
    rotation
};

std::unordered_map<tests, std::string> test_names = {
    {default_constructor, "default_constructor"},
    {four_parameter_constructor, "four_parameter_constructor"},
    {three_parameter_constructor, "three_parameter_constructor"},
    {two_parameter_constructor, "two_parameter_constructor"},
    {one_parameter_constructor, "one_parameter_constructor"},
    {unit_vector_constructor, "unit_vector_constructor"},
    {array_constructor, "array_constructor"},
    {addition, "addition"},
    {subtraction, "subtraction"},
    {multiplication, "multiplication"},
    {scalar_multiplication, "scalar_multiplication"},
    {conjugate, "conjugate"},
    {norm, "norm"},
    {inverse, "inverse"},
    {rotation, "rotation"}
};

} // namespace test_quaternion_enum
class test_quaternion_class
{
  public:
    test_quaternion_class() = default;
    ~test_quaternion_class() = default;

    std::unordered_map<test_quaternion_enum::tests, std::optional<std::string>> run()
    {
        test_addition();
        test_subtraction();
        return results;
    }

  private:
    std::unordered_map<test_quaternion_enum::tests, std::optional<std::string>> results;

    void test_constructor_default() {
        quaternion q = quaternion<int>();
        quaternion expected_result = quaternion(0, 0, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::default_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::default_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::default_constructor] = ss.str();
        }
    }

    void test_constructor_four_parameter() {
        quaternion q = quaternion<int>(1, 2, 3, 4);
        quaternion expected_result = quaternion(1, 2, 3, 4);

        if (q == expected_result)
            this->results[test_quaternion_enum::four_parameter_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::four_parameter_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::four_parameter_constructor] = ss.str();
        }
    }

    void test_constructor_three_parameter() {
        quaternion q = quaternion<int>(1, 2, 3);
        quaternion expected_result = quaternion(1, 2, 3, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::three_parameter_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::three_parameter_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::three_parameter_constructor] = ss.str();
        }
    }

    void test_constructor_two_parameter() {
        quaternion q = quaternion<int>(1, 2);
        quaternion expected_result = quaternion(1, 2, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::two_parameter_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::two_parameter_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::two_parameter_constructor] = ss.str();
        }
    }

    void test_constructor_one_parameter() {
        quaternion q = quaternion<int>(1);
        quaternion expected_result = quaternion(1, 0, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::one_parameter_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::one_parameter_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::one_parameter_constructor] = ss.str();
        }
    }

    void test_constructor_unit_vector_no_q0() {
        unit_vector<int> uv = unit_vector<int>(1, 2, 3);
        quaternion q = quaternion<int>(uv);
        quaternion expected_result = quaternion(0, 1, 2, 3);

        if (q == expected_result)
            this->results[test_quaternion_enum::unit_vector_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::unit_vector_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::unit_vector_constructor] = ss.str();
        }
    }

    void test_constructor_array() {
        quaternion q = quaternion<int>(std::array<int, 4>{1, 2, 3, 4});
        quaternion expected_result = quaternion(1, 2, 3, 4);

        if (q == expected_result)
            this->results[test_quaternion_enum::array_constructor] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::array_constructor] << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::array_constructor] = ss.str();
        }
    }

    void test_addition()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(6, 8, 10, 12);
        quaternion result = q1 + q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::addition] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::addition] << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::addition] = ss.str();
        }
    }

    void test_subtraction()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(-4, -4, -4, -4);
        quaternion result = q1 - q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::subtraction] = std::nullopt;
        else {
            std::stringstream ss;
            ss << "Testing" << test_quaternion_enum::test_names[test_quaternion_enum::subtraction] << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::subtraction] = ss.str();
        }
    }
};
} // namespace test_quaternion

i32 main()
{
    test_quaternion::test_quaternion_class test = test_quaternion::test_quaternion_class();

    std::unordered_map<test_quaternion::test_quaternion_enum::tests, std::optional<std::string>> results = test.run();

    for (auto const &result : results)
    {
        if (result.second.has_value())
            std::cout << "Test " << test_quaternion::test_quaternion_enum::test_names[result.first] << " failed: " << result.second.value() << std::endl;
        else {
            std::cout << "Test " << test_quaternion::test_quaternion_enum::test_names[result.first] << " passed" << std::endl;
        }
    }
    return 0;
}