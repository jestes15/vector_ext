#include "hip/hip_runtime.h"
#include "quaternion.cuh"
#include "types.cuh"

#include <iostream>
#include <map>
#include <optional>
#include <sstream>

namespace test_quaternion
{
namespace test_quaternion_enum
{
enum tests
{
    default_constructor,
    four_parameter_constructor,
    three_parameter_constructor,
    two_parameter_constructor,
    one_parameter_constructor,
    unit_vector_constructor,
    unit_vector_constructor_no_q0,
    classic_array_constructor,
    classic_array_constructor_no_q0,
    standard_array_constructor,
    standard_array_constructor_no_q0,
    standard_array_constructor_four,
    addition,
    subtraction,
    multiplication,
    scalar_multiplication,
    conjugate,
    norm,
    inverse,
    rotation,
    interpolate
};

std::map<tests, std::string> test_names = {{default_constructor, "default_constructor"},
                                           {four_parameter_constructor, "four_parameter_constructor"},
                                           {three_parameter_constructor, "three_parameter_constructor"},
                                           {two_parameter_constructor, "two_parameter_constructor"},
                                           {one_parameter_constructor, "one_parameter_constructor"},
                                           {unit_vector_constructor, "unit_vector_constructor"},
                                           {unit_vector_constructor_no_q0, "unit_vector_constructor_no_q0"},
                                           {classic_array_constructor, "classic_array_constructor"},
                                           {classic_array_constructor_no_q0, "classic_array_constructor_no_q0"},
                                           {standard_array_constructor, "standard_array_constructor"},
                                           {standard_array_constructor_no_q0, "standard_array_constructor_no_q0"},
                                           {standard_array_constructor_four, "standard_array_constructor_four"},
                                           {addition, "addition"},
                                           {subtraction, "subtraction"},
                                           {multiplication, "multiplication"},
                                           {scalar_multiplication, "scalar_multiplication"},
                                           {conjugate, "conjugate"},
                                           {norm, "norm"},
                                           {inverse, "inverse"},
                                           {rotation, "rotation"},
                                           {interpolate, "interpolate"}};

} // namespace test_quaternion_enum

class test_quaternion_class
{
  public:
    test_quaternion_class() = default;
    ~test_quaternion_class() = default;

    std::map<test_quaternion_enum::tests, std::optional<std::string>> run()
    {
        test_default_constructor();

        test_four_parameter_constructor();
        test_three_parameter_constructor();
        test_two_parameter_constructor();
        test_one_parameter_constructor();

        test_unit_vector_constructor();
        test_unit_vector_constructor_no_q0();

        test_classic_array_constructor();
        test_classic_array_constructor_no_q0();

        test_standard_array_constructor();
        test_standard_array_constructor_no_q0();
        test_standard_array_constructor_four();

        test_addition();
        test_subtraction();
        test_multiplication();

        test_interpolation();
        return results;
    }

  private:
    std::map<test_quaternion_enum::tests, std::optional<std::string>> results;

    void test_default_constructor()
    {
        quaternion q = quaternion();
        quaternion expected_result = quaternion(0, 0, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::default_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::default_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::default_constructor] = ss.str();
        }
    }

    void test_four_parameter_constructor()
    {
        quaternion q = quaternion(1, 2, 3, 4);
        quaternion expected_result = quaternion(1, 2, 3, 4);

        if (q == expected_result)
            this->results[test_quaternion_enum::four_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::four_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::four_parameter_constructor] = ss.str();
        }
    }

    void test_three_parameter_constructor()
    {
        quaternion q = quaternion(1, 2, 3);
        quaternion expected_result = quaternion(1, 2, 3, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::three_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::three_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::three_parameter_constructor] = ss.str();
        }
    }

    void test_two_parameter_constructor()
    {
        quaternion q = quaternion(1, 2);
        quaternion expected_result = quaternion(1, 2, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::two_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::two_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::two_parameter_constructor] = ss.str();
        }
    }

    void test_one_parameter_constructor()
    {
        quaternion q = quaternion(1);
        quaternion expected_result = quaternion(1, 0, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::one_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::one_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::one_parameter_constructor] = ss.str();
        }
    }

    void test_unit_vector_constructor()
    {
        unit_vector<int> uv = {1, 2, 3};
        quaternion q = quaternion(25, uv);
        quaternion expected_result = quaternion(25, 1, 2, 3);

        if (q == expected_result)
            this->results[test_quaternion_enum::unit_vector_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::unit_vector_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::unit_vector_constructor] = ss.str();
        }
    }

    void test_unit_vector_constructor_no_q0()
    {
        struct unit_vector<int> uv =
        {
            1, 2, 3
        };
        quaternion q = quaternion(uv);
        quaternion expected_result = quaternion(0, 1, 2, 3);

        if (q == expected_result)
            this->results[test_quaternion_enum::unit_vector_constructor_no_q0] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::unit_vector_constructor_no_q0]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::unit_vector_constructor_no_q0] = ss.str();
        }
    }

    void test_classic_array_constructor()
    {
        int arr[3] = {3, 4, 5};
        quaternion q = quaternion(91, arr);
        quaternion expected_result = quaternion(91, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::classic_array_constructor_no_q0] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::classic_array_constructor_no_q0]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::classic_array_constructor_no_q0] = ss.str();
        }
    }

    void test_classic_array_constructor_no_q0()
    {
        int arr[3] = {3, 4, 5};
        quaternion q = quaternion(arr);
        quaternion expected_result = quaternion(0, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::classic_array_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::classic_array_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::classic_array_constructor] = ss.str();
        }
    }

    void test_standard_array_constructor()
    {
        std::array<int, 3> arr = {3, 4, 5};
        quaternion q = quaternion(91, arr);
        quaternion expected_result = quaternion(91, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::standard_array_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::standard_array_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::standard_array_constructor] = ss.str();
        }
    }

    void test_standard_array_constructor_no_q0()
    {
        std::array<int, 3> arr = {3, 4, 5};
        quaternion q = quaternion(arr);
        quaternion expected_result = quaternion(0, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::standard_array_constructor_no_q0] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::standard_array_constructor_no_q0]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::standard_array_constructor_no_q0] = ss.str();
        }
    }

    void test_standard_array_constructor_four()
    {
        std::array<int, 4> arr = {2, 3, 4, 5};
        quaternion q = quaternion(arr);
        quaternion expected_result = quaternion(2, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::standard_array_constructor_four] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::standard_array_constructor_four]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::standard_array_constructor_four] = ss.str();
        }
    }

    void test_addition()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(6, 8, 10, 12);
        quaternion result = q1 + q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::addition] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::addition]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::addition] = ss.str();
        }
    }

    void test_subtraction()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(-4, -4, -4, -4);
        quaternion result = q1 - q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::subtraction] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::subtraction]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::subtraction] = ss.str();
        }
    }

    void test_multiplication()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(-60, 12, 30, 24);
        quaternion result = q1 * q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::multiplication] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::multiplication]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::multiplication] = ss.str();
        }
    }

    void test_interpolation()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion q1_normalized = q1.normalize();
        quaternion q2_normalized = q2.normalize();

        std::optional<quaternion> result =
            q1_normalized.interpolate(q2_normalized, quaternion_interpolation_method::slerp, 0.5);

        quaternion expected_result = quaternion(0.283023303767278, 0.413232827790139, 0.543442351813000, 0.673651875835861);

        if (!result.has_value())
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate]
               << " failed. Expected: " << expected_result << " Got: None";
            this->results[test_quaternion_enum::interpolate] = ss.str();
            return;
        }
        else if (result.value() == expected_result)
            this->results[test_quaternion_enum::interpolate] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate]
               << " failed. Expected: " << expected_result << " Got: " << result.value();
            this->results[test_quaternion_enum::interpolate] = ss.str();
        }
    }
};
} // namespace test_quaternion

i32 main()
{
    test_quaternion::test_quaternion_class test = test_quaternion::test_quaternion_class();

    std::map<test_quaternion::test_quaternion_enum::tests, std::optional<std::string>> results = test.run();

    for (auto const &result : results)
    {
        if (result.second.has_value())
            std::cout << "Test " << test_quaternion::test_quaternion_enum::test_names[result.first]
                      << " failed: " << result.second.value() << std::endl;
        else
        {
            std::cout << "Test " << test_quaternion::test_quaternion_enum::test_names[result.first] << " passed"
                      << std::endl;
        }
    }
    return 0;
}