#include "hip/hip_runtime.h"
#include "quaternion.cuh"
#include "types.cuh"

#include <iostream>
#include <map>
#include <optional>
#include <sstream>

namespace test_quaternion
{
namespace test_quaternion_enum
{
enum tests
{
    default_constructor,
    four_parameter_constructor,
    three_parameter_constructor,
    two_parameter_constructor,
    one_parameter_constructor,
    unit_vector_constructor,
    unit_vector_constructor_no_q0,
    classic_array_constructor,
    classic_array_constructor_no_q0,
    standard_array_constructor,
    standard_array_constructor_no_q0,
    standard_array_constructor_four,
    addition,
    subtraction,
    multiplication,
    divide,
    scalar_multiplication,
    conjugate,
    norm,
    inverse,
    rotation,
    interpolate_slerp,
    interpolate_nlerp,
    interpolate_lerp,
    normalize,
    exp,
    natural_log,
    scalar_power,
    quaternion_power,
    quaternion_rotate_not_normalized,
    quaternion_rotate_normalized,
};

std::map<tests, std::string> test_names = {
    {default_constructor, "default_constructor"},
    {four_parameter_constructor, "four_parameter_constructor"},
    {three_parameter_constructor, "three_parameter_constructor"},
    {two_parameter_constructor, "two_parameter_constructor"},
    {one_parameter_constructor, "one_parameter_constructor"},
    {unit_vector_constructor, "unit_vector_constructor"},
    {unit_vector_constructor_no_q0, "unit_vector_constructor_no_q0"},
    {classic_array_constructor, "classic_array_constructor"},
    {classic_array_constructor_no_q0, "classic_array_constructor_no_q0"},
    {standard_array_constructor, "standard_array_constructor"},
    {standard_array_constructor_no_q0, "standard_array_constructor_no_q0"},
    {standard_array_constructor_four, "standard_array_constructor_four"},
    {addition, "addition"},
    {subtraction, "subtraction"},
    {multiplication, "multiplication"},
    {divide, "divide"},
    {scalar_multiplication, "scalar_multiplication"},
    {conjugate, "conjugate"},
    {norm, "norm"},
    {inverse, "inverse"},
    {rotation, "rotation"},
    {interpolate_slerp, "interpolate_slerp"},
    {interpolate_nlerp, "interpolate_nlerp"},
    {interpolate_lerp, "interpolate_lerp"},
    {normalize, "normalize"},
    {exp, "exp"},
    {natural_log, "natural_log"},
    {scalar_power, "scalar_power"},
    {quaternion_power, "quaternion_power"},
    {quaternion_rotate_not_normalized, "quaternion_rotate_not_normalized"},
    {quaternion_rotate_normalized, "quaternion_rotate_normalized"},
};

} // namespace test_quaternion_enum

class test_quaternion_class
{
  public:
    test_quaternion_class() = default;
    ~test_quaternion_class() = default;

    std::map<test_quaternion_enum::tests, std::optional<std::string>> run()
    {
        test_default_constructor();

        test_four_parameter_constructor();
        test_three_parameter_constructor();
        test_two_parameter_constructor();
        test_one_parameter_constructor();

        test_unit_vector_constructor();
        test_unit_vector_constructor_no_q0();

        test_classic_array_constructor();
        test_classic_array_constructor_no_q0();

        test_standard_array_constructor();
        test_standard_array_constructor_no_q0();
        test_standard_array_constructor_four();

        test_addition();
        test_subtraction();
        test_multiplication();
        test_divide();

        test_interpolation_slerp();
        test_interpolation_nlerp();
        test_interpolation_lerp();

        test_normalize();
        test_conjugate();

        test_exp();
        test_natural_logarithm();
        test_scalar_power();
        test_quaternion_power();

        test_quaternion_rotate_not_normalized();

        return results;
    }

  private:
    std::map<test_quaternion_enum::tests, std::optional<std::string>> results;

    void test_default_constructor()
    {
        quaternion q = quaternion();
        quaternion expected_result = quaternion(0, 0, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::default_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::default_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::default_constructor] = ss.str();
        }
    }
    void test_four_parameter_constructor()
    {
        quaternion q = quaternion(1, 2, 3, 4);
        quaternion expected_result = quaternion(1, 2, 3, 4);

        if (q == expected_result)
            this->results[test_quaternion_enum::four_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::four_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::four_parameter_constructor] = ss.str();
        }
    }
    void test_three_parameter_constructor()
    {
        quaternion q = quaternion(1, 2, 3);
        quaternion expected_result = quaternion(1, 2, 3, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::three_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::three_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::three_parameter_constructor] = ss.str();
        }
    }
    void test_two_parameter_constructor()
    {
        quaternion q = quaternion(1, 2);
        quaternion expected_result = quaternion(1, 2, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::two_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::two_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::two_parameter_constructor] = ss.str();
        }
    }
    void test_one_parameter_constructor()
    {
        quaternion q = quaternion(1);
        quaternion expected_result = quaternion(1, 0, 0, 0);

        if (q == expected_result)
            this->results[test_quaternion_enum::one_parameter_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::one_parameter_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::one_parameter_constructor] = ss.str();
        }
    }
    void test_unit_vector_constructor()
    {
        unit_vector<int> uv = {1, 2, 3};
        quaternion q = quaternion(25, uv);
        quaternion expected_result = quaternion(25, 1, 2, 3);

        if (q == expected_result)
            this->results[test_quaternion_enum::unit_vector_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::unit_vector_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::unit_vector_constructor] = ss.str();
        }
    }
    void test_unit_vector_constructor_no_q0()
    {
        struct unit_vector<int> uv =
        {
            1, 2, 3
        };
        quaternion q = quaternion(uv);
        quaternion expected_result = quaternion(0, 1, 2, 3);

        if (q == expected_result)
            this->results[test_quaternion_enum::unit_vector_constructor_no_q0] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::unit_vector_constructor_no_q0]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::unit_vector_constructor_no_q0] = ss.str();
        }
    }
    void test_classic_array_constructor()
    {
        int arr[3] = {3, 4, 5};
        quaternion q = quaternion(91, arr);
        quaternion expected_result = quaternion(91, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::classic_array_constructor_no_q0] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::classic_array_constructor_no_q0]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::classic_array_constructor_no_q0] = ss.str();
        }
    }
    void test_classic_array_constructor_no_q0()
    {
        int arr[3] = {3, 4, 5};
        quaternion q = quaternion(arr);
        quaternion expected_result = quaternion(0, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::classic_array_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::classic_array_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::classic_array_constructor] = ss.str();
        }
    }
    void test_standard_array_constructor()
    {
        std::array<int, 3> arr = {3, 4, 5};
        quaternion q = quaternion(91, arr);
        quaternion expected_result = quaternion(91, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::standard_array_constructor] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::standard_array_constructor]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::standard_array_constructor] = ss.str();
        }
    }
    void test_standard_array_constructor_no_q0()
    {
        std::array<int, 3> arr = {3, 4, 5};
        quaternion q = quaternion(arr);
        quaternion expected_result = quaternion(0, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::standard_array_constructor_no_q0] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::standard_array_constructor_no_q0]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::standard_array_constructor_no_q0] = ss.str();
        }
    }
    void test_standard_array_constructor_four()
    {
        std::array<int, 4> arr = {2, 3, 4, 5};
        quaternion q = quaternion(arr);
        quaternion expected_result = quaternion(2, 3, 4, 5);

        if (q == expected_result)
            this->results[test_quaternion_enum::standard_array_constructor_four] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::standard_array_constructor_four]
               << " failed. Expected: " << expected_result << " Got: " << q;
            this->results[test_quaternion_enum::standard_array_constructor_four] = ss.str();
        }
    }
    void test_addition()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(6, 8, 10, 12);
        quaternion result = q1 + q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::addition] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::addition]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::addition] = ss.str();
        }
    }
    void test_subtraction()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(-4, -4, -4, -4);
        quaternion result = q1 - q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::subtraction] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::subtraction]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::subtraction] = ss.str();
        }
    }
    void test_multiplication()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(-60, 12, 30, 24);
        quaternion result = q1 * q2;

        if (result == expected_result)
            this->results[test_quaternion_enum::multiplication] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::multiplication]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::multiplication] = ss.str();
        }
    }
    void test_interpolation_slerp()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion q1_normalized = q1.normalize();
        quaternion q2_normalized = q2.normalize();

        std::optional<quaternion> result =
            q1_normalized.interpolate(q2_normalized, quaternion_interpolation_method::slerp, 0.5);

        quaternion expected_result =
            quaternion(0.283023303767278, 0.413232827790139, 0.543442351813000, 0.673651875835861);

        if (!result.has_value())
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate_slerp]
               << " failed. Expected: " << expected_result << " Got: None";
            this->results[test_quaternion_enum::interpolate_slerp] = ss.str();
            return;
        }
        else if ((result.value() - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::interpolate_slerp] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate_slerp]
               << " failed. Expected: " << expected_result << " Got: " << result.value();
            this->results[test_quaternion_enum::interpolate_slerp] = ss.str();
        }
    }
    void test_interpolation_nlerp()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion q1_normalized = q1.normalize();
        quaternion q2_normalized = q2.normalize();

        std::optional<quaternion> result =
            q1_normalized.interpolate(q2_normalized, quaternion_interpolation_method::nlerp, 0.5);

        quaternion expected_result =
            quaternion(0.283023303767278, 0.413232827790139, 0.543442351813000, 0.673651875835862);

        if (!result.has_value())
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate_nlerp]
               << " failed. Expected: " << expected_result << " Got: None";
            this->results[test_quaternion_enum::interpolate_nlerp] = ss.str();
            return;
        }
        else if ((result.value() - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::interpolate_nlerp] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate_nlerp]
               << " failed. Expected: " << expected_result << " Got: " << result.value();
            this->results[test_quaternion_enum::interpolate_nlerp] = ss.str();
        }
    }
    void test_interpolation_lerp()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion q1_normalized = q1.normalize();
        quaternion q2_normalized = q2.normalize();

        std::optional<quaternion> result =
            q1_normalized.interpolate(q2_normalized, quaternion_interpolation_method::lerp, 0.5);

        quaternion expected_result =
            quaternion(0.280811603812254, 0.410003598908726, 0.539195594005199, 0.668387589101672);

        if (!result.has_value())
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate_lerp]
               << " failed. Expected: " << expected_result << " Got: None";
            this->results[test_quaternion_enum::interpolate_lerp] = ss.str();
            return;
        }
        else if ((result.value() - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::interpolate_lerp] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::interpolate_lerp]
               << " failed. Expected: " << expected_result << " Got: " << result.value();
            this->results[test_quaternion_enum::interpolate_lerp] = ss.str();
        }
    }
    void test_normalize()
    {
        quaternion q(1, 2, 3, 4);

        quaternion result = q.normalize();

        quaternion expected_result =
            quaternion(0.182574185835055, 0.365148371670111, 0.547722557505166, 0.730296743340221);

        if ((result - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::normalize] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::normalize]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::normalize] = ss.str();
        }
    }
    void test_conjugate()
    {
        quaternion q(1, 2, 3, 4);

        quaternion result = q.complex_conjugate();

        quaternion expected_result = quaternion(1, -2, -3, -4);

        if (result == expected_result)
            this->results[test_quaternion_enum::conjugate] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::conjugate]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::conjugate] = ss.str();
        }
    }
    void test_divide()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion result = q1 / q2;

        quaternion expected_result = quaternion(0.402298850574713, 0.0, 0.091954022988506, 0.045977011494253);

        if ((result - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::divide] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::divide]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::divide] = ss.str();
        }
    }
    void test_exp()
    {
        quaternion q = quaternion(1, 2, 3, 4);

        quaternion result = q.exp();
        quaternion expected_result =
            quaternion(1.693922723683299, -0.789559624541559, -1.184339436812338, -1.579119249083118);

        if ((result - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::exp] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::exp]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::exp] = ss.str();
        }
    }
    void test_natural_logarithm()
    {
        quaternion q = quaternion(1, 2, 3, 4);

        quaternion result = q.ln();
        quaternion expected_result =
            quaternion(1.7005986908311, 0.515190292664085, 0.772785438996128, 1.030380585328170);

        if ((result - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::natural_log] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::natural_log]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::natural_log] = ss.str();
        }
    }
    void test_scalar_power()
    {
        quaternion q = quaternion(1, 2, 3, 4);

        quaternion result = q.pow(2);
        quaternion expected_result = quaternion(-28, 4, 6, 8);

        if ((result - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::scalar_power] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::scalar_power]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::scalar_power] = ss.str();
        }
    }
    void test_quaternion_power()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion result = q1.pow(q2);
        quaternion expected_result = quaternion(-0.000228445, 5.57689e-05, 8.46271e-05, 8.41402e-05);

        if ((result - expected_result).abs() < this->quaternion_epsilon)
            this->results[test_quaternion_enum::quaternion_power] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::quaternion_power]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::quaternion_power] = ss.str();
        }
    }

    void test_quaternion_rotate_not_normalized()
    {
        quaternion q = quaternion(1, 2, 3, 4);
        rotation_vector<double> v = rotation_vector<double>{1.0, 2.0, 3.0};

        rotation_vector<double> result = q.rotate(v);
        rotation_vector<double> expected_result =
            rotation_vector<double>{1.666666666666667, 2.266666666666667, 2.466666666666667};

        if (abs(result - expected_result) < this->rotation_epsilon)
            this->results[test_quaternion_enum::quaternion_rotate_not_normalized] = std::nullopt;
        else
        {
            std::stringstream ss;
            ss << "Testing " << test_quaternion_enum::test_names[test_quaternion_enum::quaternion_rotate_not_normalized]
               << " failed. Expected: " << expected_result << " Got: " << result;
            this->results[test_quaternion_enum::quaternion_rotate_not_normalized] = ss.str();
        }
    }
    quaternion quaternion_epsilon = quaternion(0.00001, 0.00001, 0.00001, 0.00001);
    rotation_vector<double> rotation_epsilon = rotation_vector<double>{0.00001, 0.00001, 0.00001};
};
} // namespace test_quaternion

i32 main()
{
    test_quaternion::test_quaternion_class test = test_quaternion::test_quaternion_class();
    std::map<test_quaternion::test_quaternion_enum::tests, std::optional<std::string>> results = test.run();

    for (auto const &result : results)
    {
        if (result.second.has_value())
            std::cout << "Test " << test_quaternion::test_quaternion_enum::test_names[result.first]
                      << " failed: " << result.second.value() << std::endl;
        else
        {
            std::cout << "Test " << test_quaternion::test_quaternion_enum::test_names[result.first] << " passed"
                      << std::endl;
        }
    }
    return 0;
}