#include "hip/hip_runtime.h"
#include "quaternion.cuh"
#include "types.cuh"

#include <iostream>
#include <unordered_map>

namespace test_quaternion
{
namespace test_quaternion_enum
{
enum tests
{
    addition,
    multiplication,
    scalar_multiplication,
    conjugate,
    norm,
    inverse,
    rotation
};
} // namespace test_quaternion_enum
class test_quaternion_class
{
  public:
    test_quaternion_class() = default;
    ~test_quaternion_class() = default;

    std::unordered_map<test_quaternion_enum::tests, std::pair<i64, std::string>> run()
    {
        test_addition();
        return results;
    }

  private:
    std::unordered_map<test_quaternion_enum::tests, std::pair<i64, std::string>> results;

    void test_addition()
    {
        quaternion q1(1, 2, 3, 4);
        quaternion q2(5, 6, 7, 8);

        quaternion expected_result = quaternion(6, 8, 10, 12);
		quaternion result = q1 + q2;

		if (result == expected_result)
			this->results[test_quaternion_enum::addition] = std::make_pair(0, "Test Passed");
		else
            this->results[test_quaternion_enum::addition] = std::make_pair(1, "Test Failed");
    }
};
} // namespace test_quaternion

i32 main()
{
    test_quaternion::test_quaternion_class test = test_quaternion::test_quaternion_class();

    std::unordered_map<test_quaternion::test_quaternion_enum::tests, std::pair<i64, std::string>> results = test.run();
    return 0;
}