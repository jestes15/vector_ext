﻿#include <chrono>
#include <iostream>
#include <vector>

#include "hip/hip_runtime.h"
#include "hiprand.h"

#include "hip/hip_runtime.h"
#include ""

#define USE_CUDA
#define MATRIX_MUL
#include "kernel_impl.cuh"
#include "vector_ext.cuh"

#include <unordered_set>

template <typename T> void print_array(std::vector<T> &array)
{
    std::cout << "[ ";
    for (auto &i : array)
    {
        std::cout << i << " ";
    }
    std::cout << "]" << std::endl;
}

template <typename T>
int validate_dest(std_vec::vector_ext<T> &dest, std_vec::vector_ext<T> &src, std_vec::vector_ext<T> &src2)
{
    for (int i = 0; i < dest.size(); i++)
    {
        if (dest[i] != src[i] + src2[i])
        {
            std::cout << "Error at index " << i << std::endl;
            return 1;
        }
    }
    return 0;
}

namespace tests
{
int something()
{
    int iLen(1024);
    long long *device_result;
    long long result[iLen] = {0};

    CUDA_CALL(hipMalloc(reinterpret_cast<long long **>(&device_result), sizeof(long long) * iLen));

    CUDA_CALL(hipMemcpy(device_result, result, sizeof(long long) * iLen, hipMemcpyHostToDevice));

    dim3 block(iLen);
    dim3 grid((iLen + block.x - 1) / block.x);

    auto start =
        std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    std_vec::kernel::mystery_kernel<<<grid, block>>>(device_result);
    auto stop =
        std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();

    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(result, device_result, sizeof(long long) * iLen, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(device_result));

    std::cout << stop - start << std::endl;

    return EXIT_SUCCESS;
}

#if defined(MATRIX_MUL)
// TODO: Finish test for user_space::matrix_mul
void _test()
{
    const int size = 4;

    auto src = new int[size * size];
    auto src2 = new int[size * size];
    auto dest = new int[size * size];

    std::random_device gen;
    std::uniform_int_distribution<int> dist(0, 20);

    std::for_each_n(src, size * size, [&dist, &gen](int &i) { i = static_cast<int>(dist(gen)); });

    std::for_each_n(src2, size * size, [&dist, &gen](int &i) { i = static_cast<int>(dist(gen)); });

    for (unsigned long long i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%d\t", src[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (unsigned long long i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%d\t", src2[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");

    std_vec::user_space::matrix_mul(dest, src, src2, size, size, size, size);

    for (unsigned long long i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%d\t", dest[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");
}
#endif

int t()
{
    long size = 6000000;
    std_vec::vector_ext<int> src1(size), src2(size), dest(size);

    for (auto i = 0; i < size; ++i)
    {
        src1.push_back(i);
        src2.push_back(i);
    }

    auto start_for =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    for (long i = 0; i < size; ++i)
    {
        dest[i] = src1[i] + src2[i];
    }
    auto end_for =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    std::cout << "Time taken: " << (end_for - start_for) << "ms" << std::endl;

    auto start =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    dest = src1 + src2;
    auto stop =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();

    std::cout << "Time taken: " << (stop - start) << "ms" << std::endl;

    if (validate_dest(dest, src1, src2))
    {
        return 1;
    }

    std_vec::vector_ext<int> src(20);
    src.generate_random_list_cuda(10000);

    print_array(src);

    return 0;
}
} // namespace tests

int main()
{
    #ifdef USE_CUDA
    tests::_test();
    #endif

    return 0;
}