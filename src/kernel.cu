﻿#include <algorithm>
#include <chrono>
#include <iostream>
#include <vector>

#include "hip/hip_runtime.h"
#include "hiprand.h"

#include "hip/hip_runtime.h"
#include ""

#define USE_CUDA
#define MATRIX_MUL
#include "kernel_impl.cuh"
#include "vector_ext.cuh"

template <typename T> void print_array(std::vector<T> &array)
{
    std::cout << "[ ";
    for (auto &i : array)
    {
        std::cout << i << " ";
    }
    std::cout << "]" << std::endl;
}

template <typename T>
i32 validate_dest(std_vec::vector_ext<T> &dest, std_vec::vector_ext<T> &src, std_vec::vector_ext<T> &src2)
{
    for (i32 i = 0; i < dest.size(); i++)
    {
        if (dest[i] != src[i] + src2[i])
        {
            std::cout << "Error at index " << i << std::endl;
            return 1;
        }
    }
    return 0;
}

namespace tests
{
#if defined(MATRIX_MUL)
// TODO: Finish test for user_space::matrix_mul
void _test()
{
    const i32 size = 4;

    auto src = new int[size * size];
    auto src2 = new int[size * size];
    auto dest = new int[size * size];

    std::random_device gen;
    std::uniform_int_distribution<int> dist(0, 20);

    std::for_each_n(src, size * size, [&dist, &gen](i32 &i) { i = static_cast<int>(dist(gen)); });

    std::for_each_n(src2, size * size, [&dist, &gen](i32 &i) { i = static_cast<int>(dist(gen)); });

    for (unsigned long long i = 0; i < size; i++)
    {
        for (i32 j = 0; j < size; j++)
        {
            printf("%d\t", src[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (unsigned long long i = 0; i < size; i++)
    {
        for (i32 j = 0; j < size; j++)
        {
            printf("%d\t", src2[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");

    std_vec::user_space::matrix_mul(dest, src, src2, size, size, size, size);

    for (unsigned long long i = 0; i < size; i++)
    {
        for (i32 j = 0; j < size; j++)
        {
            printf("%d\t", dest[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");
}
#endif

i32 t()
{
    long size = 6000000;
    std_vec::vector_ext<int> src1(size), src2(size), dest(size);

    for (auto i = 0; i < size; ++i)
    {
        src1.push_back(i);
        src2.push_back(i);
    }

    auto start_for =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    for (long i = 0; i < size; ++i)
    {
        dest[i] = src1[i] + src2[i];
    }
    auto end_for =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    std::cout << "Time taken: " << (end_for - start_for) << "ms" << std::endl;

    auto start =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();
    dest = src1 + src2;
    auto stop =
        std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())
            .count();

    std::cout << "Time taken: " << (stop - start) << "ms" << std::endl;

    if (validate_dest(dest, src1, src2))
    {
        return 1;
    }

    std_vec::vector_ext<int> src(20);
    src.generate_random_list_cuda(10000);

    print_array(src);

    return 0;
}
} // namespace tests

i32 main()
{
    #ifdef USE_CUDA
    tests::_test();
    #endif

    return 0;
}